#include "hip/hip_runtime.h"
/*
 * CUDABrownianThermostat.cpp
 *
 *  Created on: Feb 15, 2013
 *      Author: rovigatti
 */

#include <hiprand/hiprand_kernel.h>

#include "CUDABrownianThermostat.h"

__global__ void brownian_thermostat(hiprandState *rand_state, number4 *vels, number4 *Ls, number rescale_factor, number pt, number pr, int N) {
	if(IND < N) {
		hiprandState state = rand_state[IND];

		if(hiprand_uniform(&state) < pt) {
			number4 v;
			number trash;

			gaussian(state, v.x, v.y);
			gaussian(state, v.z, trash);

			v.x *= rescale_factor;
			v.y *= rescale_factor;
			v.z *= rescale_factor;
			v.w = (v.x * v.x + v.y * v.y + v.z * v.z) * (number) 0.5f;

			vels[IND] = v;
		}

		if(hiprand_uniform(&state) < pr) {
			number4 L;
			number trash;

			gaussian(state, L.x, L.y);
			gaussian(state, L.z, trash);

			L.x *= rescale_factor;
			L.y *= rescale_factor;
			L.z *= rescale_factor;
			L.w = (L.x * L.x + L.y * L.y + L.z * L.z) * (number) 0.5f;

			Ls[IND] = L;
		}

		rand_state[IND] = state;
	}
}

CUDABrownianThermostat::CUDABrownianThermostat() :
				CUDABaseThermostat(),
				BrownianThermostat() {

}

CUDABrownianThermostat::~CUDABrownianThermostat() {

}

void CUDABrownianThermostat::get_settings(input_file &inp) {
	BrownianThermostat::get_settings(inp);
	CUDABaseThermostat::get_cuda_settings(inp);
}

void CUDABrownianThermostat::init(int N) {
	BrownianThermostat::init(N);

	this->_setup_rand(N);
}

bool CUDABrownianThermostat::would_activate(llint curr_step) {
	return (curr_step % this->_newtonian_steps == 0);
}

void CUDABrownianThermostat::apply_cuda(number4 *d_poss, GPU_quat *d_orientations, number4 *d_vels, number4 *d_Ls, llint curr_step) {
	if(!would_activate(curr_step)) return;

brownian_thermostat
<<<this->_launch_cfg.blocks, this->_launch_cfg.threads_per_block>>>
(this->_d_rand_state, d_vels, d_Ls, this->_rescale_factor, this->_pt, this->_pr, this->_N_part);
}
